//
// Created by Soeren Peters on 15.12.20.
//

#include "Cuda_example.h"

#include <sstream>
#include <string>

#include <hip/hip_runtime_api.h>

#include <spdlog/spdlog.h>

#include "device/kernel.cuh"

namespace cpp_start
{

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

std::vector<int> Cuda_example::add(const std::vector<int> &a, const std::vector<int> &b) const
{
    const auto size = a.size();
    std::vector<int> result(size);

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(result.data(), a.data(), b.data(), (unsigned int)size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return {};
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return {};
    }

    return result;
}

int Cuda_example::getNumberOfDevices() const
{
    int nDevices;

    hipGetDeviceCount(&nDevices);

    return nDevices;
}

std::string Cuda_example::getDeviceName(int deviceNumber) const
{
    checkDeviceNumber(deviceNumber);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNumber);

    return { prop.name };
}

std::string Cuda_example::getComputeCapability(int deviceNumber) const
{
    checkDeviceNumber(deviceNumber);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNumber);

    std::stringstream cc;
    cc << prop.major << "." << prop.minor;

    return cc.str();
}

size_t Cuda_example::getTotalGlobalMemory(int deviceNumber) const
{
    checkDeviceNumber(deviceNumber);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNumber);

    return prop.totalGlobalMem;
}

double Cuda_example::getPeakMemoryBandwidth(int deviceNumber) const
{
    checkDeviceNumber(deviceNumber);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNumber);

    return 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6;
}

void Cuda_example::checkDeviceNumber(int deviceNumber) const
{
    const auto numberOfDevices = getNumberOfDevices();

    std::stringstream error_message;
    error_message << "Device number " << deviceNumber << "not valid. Number of devices: " << numberOfDevices;

    if (deviceNumber >= numberOfDevices)
        throw std::runtime_error(error_message.str());
}


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void **)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    add_cuda_kernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

} // namespace cpp_start

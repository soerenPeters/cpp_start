#include "hip/hip_runtime.h"
#include "kernel.cuh"

#include <hip/hip_runtime_api.h>
#include <>


__global__ void add_cuda_kernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i]  = a[i] + b[i];
}
